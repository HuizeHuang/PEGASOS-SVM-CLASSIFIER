#include "hip/hip_runtime.h"
#include <random>
#include <vector>
#include <iostream>
#include <string>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/copy.h>
#include <thrust/fill.h>

#include "svm_classifier.hpp"

using namespace std;

SVMClassifier::SVMClassifier(double c, unsigned int epochs, unsigned int seed, unsigned int feature_size, unsigned int batch_size) {
  this->c = c;
  this->epochs = epochs;
  this->seed = seed;
  this->feature_size = feature_size;
  this->batch_size = batch_size;
}

__global__ void predict_array(KernelArray<double> kArray, double *w, int * d_y_pred, unsigned int feature_size) {
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    
    double dot_product = 0;
    
    for(int j = 0; j < feature_size; j++) {
        dot_product += w[j]*kArray._array[i*feature_size + j];
    }

    if (dot_product >= 0) {
        d_y_pred[i] = 1;
    }

    else d_y_pred[i] = -1;
}

__global__ void copy_batch(KernelArray<double> kArray, double *xi, unsigned int idx, unsigned int feature_size, unsigned int batch_size) {
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    if(i < feature_size*batch_size) {
        xi[i] = kArray._array[idx*feature_size+i];
    }
}

__global__ void copy_array(double*a, double *b, unsigned int feature_size) {
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    if(i < feature_size) {
        a[i] = b[i];
    }
    
}

__global__ void select_samples(double *xi, double *w, unsigned int feature_size, unsigned int idx, KernelArray<int> label, unsigned int batch_size) {
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    if(i < batch_size) {
        double dot_product = 0;
    
        for(int j = 0; j < feature_size; j++) {
            dot_product += w[j]*xi[i*feature_size + j];
        }

        if(dot_product*label._array[idx+i] >= 1) {
            for(int j = 0; j < feature_size; j++) {
                xi[i*feature_size + j] = 0;
            }
        }
    }
    
}

__global__ void reduce_by_samples(double *yi_xi, double *xi, unsigned int batch_size, unsigned int feature_size, unsigned int idx, KernelArray<int> label) {
    int i=blockIdx.x*blockDim.x+threadIdx.x;

    if(i < feature_size) {
        double sum = 0;
    
        for(int j = 0; j < batch_size; j++) {
            sum += xi[j*feature_size + i]*label._array[idx+j]; //acho que esta OK
        }

        yi_xi[i] = sum;
    }
    
}

__global__ void update_w(double *yi_xi, double *w, double *next_w, double nt, double c, int batch_size, unsigned int feature_size) {

    int i=blockIdx.x*blockDim.x+threadIdx.x;
    if (i < feature_size) {
        next_w[i] = w[i] - nt*c*w[i] + (nt/batch_size)*yi_xi[i];
    } 
}

void SVMClassifier::fit(thrust::device_vector<double> & data, thrust::device_vector<int> & label) {
    srand(seed);  
    
    hipMalloc((void **)&w, feature_size*sizeof(double));
    hipMemset(w, 0, feature_size*sizeof(double));

    double * xi;
    hipMalloc((void **)&xi, batch_size*feature_size*sizeof(double));

    double * yi_xi;
    hipMalloc((void **)&yi_xi, feature_size*sizeof(double));

    double * next_w;
    hipMalloc((void **)&next_w, feature_size*sizeof(double));

    //convert device_vector to gpu array
    KernelArray<double> ka_data = convertToKernel(data);
    KernelArray<int> ka_label = convertToKernel(label);

    for(unsigned int t = 1; t < epochs; t++) {
        hipMemset(next_w, 0, feature_size*sizeof(double));

        double nt = 1/(c*t);    
        unsigned int idx = rand() % (label.size() - batch_size);


        copy_batch<<<ceil(batch_size*feature_size/512.0), 512>>>(ka_data, xi, idx, feature_size, batch_size);
        

        select_samples<<<ceil(batch_size/512.0), 512>>>(xi, w, feature_size, idx, ka_label, batch_size);


        reduce_by_samples<<<ceil(feature_size/512.0), 512>>>(yi_xi, xi, batch_size, feature_size, idx, ka_label);


        update_w<<<ceil(feature_size/512.0), 512>>>(yi_xi, w, next_w, nt, c, batch_size, feature_size);


        copy_array<<<ceil(feature_size/512.0), 512>>>(w, next_w, feature_size);

    }

    // double *h_b;
    // h_b = (double *)malloc(feature_size*sizeof(double));
    // hipMemcpy(h_b, w, feature_size*sizeof(double), hipMemcpyDeviceToHost);
    // for(unsigned int i = 0; i < feature_size; i++) {
    //     cout << "w["<<i<<"] = " << h_b[i] << endl;
    // }

    hipFree(xi);
    hipFree(yi_xi);
    hipFree(next_w);

    cout << endl;
}

thrust::device_vector<int> SVMClassifier::predict(thrust::device_vector<double> & data) {
    thrust::device_vector<int> predicted_labels;
    
    int *d_predicted_labels;
    hipMalloc((void **)&d_predicted_labels, data.size()/feature_size*sizeof(int));


    KernelArray<double> ka_data = convertToKernel(data);

    predict_array<<<ceil((data.size()/feature_size)/512.0), 512>>>(ka_data, w, d_predicted_labels, feature_size);

    int *y_pred;
    y_pred = (int *)malloc(data.size()/feature_size*sizeof(int));
    hipMemcpy(y_pred, d_predicted_labels, data.size()/feature_size*sizeof(int), hipMemcpyDeviceToHost);

    for(unsigned int i = 0; i < data.size()/feature_size; i++) {
        predicted_labels.push_back(y_pred[i]);
    }

    hipFree(w);
    hipFree(d_predicted_labels);
    free(y_pred);


    return predicted_labels;
}

double SVMClassifier::accuracy(thrust::device_vector<int> & label, thrust::device_vector<int> & pred_label) {
    int correct_pred = 0;

    
    for(unsigned int i = 0; i < label.size(); i++) {
        if (label[i] == pred_label[i]) {
            correct_pred += 1;
        }
    }

    return (double) correct_pred/label.size();
}

