#include <random>
#include <vector>
#include <iostream>
#include <string>
#include <time.h>
#include <algorithm>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>

#include "svm_classifier.hpp"
#include "data_reader.hpp"

using namespace std;

void set_envs(string & data_path, double *c, unsigned int *samples_limit, unsigned int *epochs, unsigned int *batch_size, unsigned int *train_size, unsigned int *num_iterations) {
    
    char *data_path_env, *c_env, *samples_limit_env, *epochs_env, *batch_size_env, *train_size_env, *num_iterations_env;

    data_path_env = getenv ("DATA_PATH");
    if(data_path_env != NULL) {
        data_path = data_path_env;
    }

    c_env = getenv ("C");
    if(c_env != NULL) {
        *c = atof(c_env);
    }
     
    samples_limit_env = getenv ("SAMPLES_LIMIT");
    if(samples_limit_env != NULL) {
        *samples_limit = atoi(samples_limit_env);
    }

    epochs_env = getenv ("EPOCHS");
    if(epochs_env != NULL) {
        *epochs = atoi(epochs_env);
    }

    batch_size_env = getenv ("BATCH_SIZE");
    if(batch_size_env != NULL) {
        *batch_size = atoi(batch_size_env);
    }

    train_size_env = getenv ("TRAIN_SIZE");
    if(train_size_env != NULL) {
        *train_size = 1/(1-atof(train_size_env));
    }

    num_iterations_env = getenv ("NUM_ITERATIONS");
    if(num_iterations_env != NULL) {
        *num_iterations = atoi(num_iterations_env);
    }
}

int main(int argc, char *argv[]) {
    hipEvent_t start, stop, iter_start, iter_stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&iter_start);
    hipEventCreate(&iter_stop);

    string data_path = "../datasets/diabetes.csv";
    double c = 0.001;
    unsigned int samples_limit = 9999999;
    unsigned int epochs = 100000;
    unsigned int batch_size = 10;
    unsigned int train_size = 5;
    unsigned int num_iterations = 5;

    set_envs(data_path, &c, &samples_limit, &epochs, &batch_size, &train_size, &num_iterations);

    cout << "Reading and parsing data: " << endl;

    vector<vector<double>> h_data = read_data(data_path.c_str(), samples_limit);
    
    const unsigned int f_size = feature_size(h_data);
    
    random_shuffle(h_data.begin(), h_data.end());

    vector<double> h_labels = set_labels(h_data);

    cout << "Done " << endl;

    cout << "Sending data to gpu: " << endl;
    
    thrust::device_vector<double> data;

    for (vector<double> sample: h_data) {
        thrust::device_vector<double> aux(sample);
        data.insert(data.end(), aux.begin(), aux.end());   
    }

    thrust::device_vector<double> labels(h_labels);

    thrust::device_vector<double> x_test(data.begin(), data.begin() + (data.size()/train_size) - ((data.size()/train_size)%f_size));
    thrust::device_vector<double> x_train(data.begin() + (data.size()/train_size) - ((data.size()/train_size)%f_size), data.end());

    thrust::device_vector<int> y_test(labels.begin(), labels.begin() + labels.size()/train_size);
    thrust::device_vector<int> y_train(labels.begin() + labels.size()/train_size, labels.end());

    cout << "Done " << endl;
    
    // cout << x_train.size() << endl;
    // cout << y_train.size() << endl;

    double total_acc = 0;

    thrust::device_vector<int> y_pred(y_test.size());
    
    hipEventRecord(start, NULL);

    cout << "Started SVM: " << endl;

    for (unsigned int i = 0; i < num_iterations; i++) {

        

        SVMClassifier* svm_clf = new SVMClassifier(c, epochs, time(NULL)+i, f_size, batch_size);
        //cout << "seed: " << time(NULL) << endl;

        hipEventRecord(iter_start, NULL);

        // cout << "Fitting data: " << endl;
        svm_clf->fit(x_train, y_train);
        // cout << "Done " << endl;

        hipEventRecord(iter_stop, NULL);
        hipEventSynchronize(iter_stop);
        float iter_msecTotal = 0.0f;
        hipEventElapsedTime(&iter_msecTotal, iter_start, iter_stop);
        cout << "Current iteration training time: " << iter_msecTotal/1000 << " seconds" << endl;

        // cout << "Predicting data: " << endl;
        y_pred = svm_clf->predict(x_test);
        // cout << "Done " << endl;

        double cur_acc = svm_clf->accuracy(y_test, y_pred);

        total_acc += cur_acc;

        cout << "accuracy: "<< cur_acc  << endl;
        
        

       

    }

    cout << endl << "mean accuracy: "<< total_acc/num_iterations  << endl;

    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);

    float msecTotal = 0.0f;
    hipEventElapsedTime(&msecTotal, start, stop);
    cout << "Elapsed total training + predicting time: " << msecTotal/1000 << " seconds" << endl;

    return 0;
}